
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.60
Copyright (C) 2018  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA

#include <cuda.h>
#include <cuda_runtime.h>
#include "BoxDimensions.h"
#include "CalculateEwaldCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "CUDAMemoryManager.cuh"
#include "cub/cub.cuh"
#include <vector>

using namespace cub;

#define FULL_MASK 0xffffffff

void CallBoxReciprocalSetupGPU(VariablesCUDA *vars,
                               XYZArray const &coords,
                               double const *kx,
                               double const *ky,
                               double const *kz,
                               std::vector<double> particleCharge,
                               uint imageSize,
                               double *sumRnew,
                               double *sumInew,
                               double *prefact,
                               double *hsqr,
                               double &energyRecip,
                               uint box)
{
  double *gpu_particleCharge;
  double * gpu_energyRecip;
  double * gpu_final_energyRecip;
  int blocksPerGrid, threadsPerBlock;
  int atomNumber = coords.Count();

  CUMALLOC((void**) &gpu_particleCharge,
           particleCharge.size() * sizeof(double));
  CUMALLOC((void**) &gpu_energyRecip, imageSize * sizeof(double));
  CUMALLOC((void**) &gpu_final_energyRecip, sizeof(double));

  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_kx[box], kx, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_ky[box], ky, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_kz[box], kz, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_prefact[box], prefact, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_hsqr[box], hsqr, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);

  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  BoxReciprocalSetupGPU <<< blocksPerGrid, threadsPerBlock>>>(
    vars->gpu_x,
    vars->gpu_y,
    vars->gpu_z,
    vars->gpu_kx[box],
    vars->gpu_ky[box],
    vars->gpu_kz[box],
    atomNumber,
    gpu_particleCharge,
    vars->gpu_sumRnew[box],
    vars->gpu_sumInew[box],
    imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  BoxReciprocalGPU <<< blocksPerGrid, threadsPerBlock>>>(
    vars->gpu_prefact[box],
    vars->gpu_sumRnew[box],
    vars->gpu_sumInew[box],
    gpu_energyRecip,
    imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  cudaMemcpy(sumRnew, vars->gpu_sumRnew[box],
             imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
  cudaMemcpy(sumInew, vars->gpu_sumInew[box],
             imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);

  // ReduceSum
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecip,
                    gpu_final_energyRecip, imageSize);
  CUMALLOC(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecip,
                    gpu_final_energyRecip, imageSize);
  cudaMemcpy(&energyRecip, gpu_final_energyRecip,
             sizeof(double), cudaMemcpyDeviceToHost);

  CUFREE(gpu_particleCharge);
  CUFREE(gpu_energyRecip);
  CUFREE(gpu_final_energyRecip);
  CUFREE(d_temp_storage);
}

void CallMolReciprocalGPU(VariablesCUDA *vars,
                          XYZArray const &currentCoords,
                          XYZArray const &newCoords,
                          std::vector<double> particleCharge,
                          uint imageSize,
                          double *sumRnew,
                          double *sumInew,
                          double &energyRecipNew,
                          uint box)
{
  // Calculate atom number
  int atomNumber = currentCoords.Count();
  int newCoordsNumber = newCoords.Count();
  double *gpu_particleCharge;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_energyRecipNew, *gpu_final_energyRecipNew;

  CUMALLOC((void**) &gpu_particleCharge,
           particleCharge.size() * sizeof(double));
  CUMALLOC((void**) &gpu_energyRecipNew, imageSize * sizeof(double));
  CUMALLOC((void**) &gpu_final_energyRecipNew, sizeof(double));

  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_nx, newCoords.x, newCoordsNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_ny, newCoords.y, newCoordsNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_nz, newCoords.z, newCoordsNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  MolReciprocalGPU <<< blocksPerGrid,
                   threadsPerBlock>>>(vars->gpu_x, vars->gpu_y, vars->gpu_z,
                                      vars->gpu_nx, vars->gpu_ny, vars->gpu_nz,
                                      vars->gpu_kxRef[box], vars->gpu_kyRef[box],
                                      vars->gpu_kzRef[box],
                                      atomNumber,
                                      gpu_particleCharge,
                                      vars->gpu_sumRnew[box],
                                      vars->gpu_sumInew[box],
                                      vars->gpu_sumRref[box],
                                      vars->gpu_sumIref[box],
                                      vars->gpu_prefactRef[box],
                                      gpu_energyRecipNew,
                                      imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  cudaMemcpy(sumRnew, vars->gpu_sumRnew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
  cudaMemcpy(sumInew, vars->gpu_sumInew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);

  // ReduceSum
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  CUMALLOC(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  cudaMemcpy(&energyRecipNew, gpu_final_energyRecipNew,
             sizeof(double), cudaMemcpyDeviceToHost);


  CUFREE(gpu_particleCharge);
  CUFREE(gpu_energyRecipNew);
  CUFREE(gpu_final_energyRecipNew);
  CUFREE(d_temp_storage);
}

void CallSwapReciprocalGPU(VariablesCUDA *vars,
                           XYZArray const &coords,
                           std::vector<double> particleCharge,
                           uint imageSize,
                           double *sumRnew,
                           double *sumInew,
                           int const insert,
                           double &energyRecipNew,
                           uint box)
{
  // Calculate atom number
  int atomNumber = coords.Count();
  // given coordinates
  double *gpu_particleCharge;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_energyRecipNew, *gpu_final_energyRecipNew;

  CUMALLOC((void**) &gpu_particleCharge,
           particleCharge.size() * sizeof(double));
  CUMALLOC((void**) &gpu_energyRecipNew, imageSize * sizeof(double));
  CUMALLOC((void**) &gpu_final_energyRecipNew, sizeof(double));

  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  SwapReciprocalGPU <<< blocksPerGrid,
                    threadsPerBlock>>>(vars->gpu_x, vars->gpu_y, vars->gpu_z,
                                       vars->gpu_kxRef[box], vars->gpu_kyRef[box],
                                       vars->gpu_kzRef[box],
                                       atomNumber,
                                       gpu_particleCharge,
                                       vars->gpu_sumRnew[box],
                                       vars->gpu_sumInew[box],
                                       vars->gpu_sumRref[box],
                                       vars->gpu_sumIref[box],
                                       vars->gpu_prefactRef[box],
                                       insert,
                                       gpu_energyRecipNew,
                                       imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);
//#ifndef NDEBUG
  // In the future maybe we could remove this for Nondebug?
  cudaMemcpy(sumRnew, vars->gpu_sumRnew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
  cudaMemcpy(sumInew, vars->gpu_sumInew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
//#endif

  // ReduceSum
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  CUMALLOC(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  cudaMemcpy(&energyRecipNew, gpu_final_energyRecipNew,
             sizeof(double), cudaMemcpyDeviceToHost);

  CUFREE(gpu_particleCharge);
  CUFREE(gpu_energyRecipNew);
  CUFREE(gpu_final_energyRecipNew);
  CUFREE(d_temp_storage);
}

void CallBoxForceReciprocalGPU(
  VariablesCUDA *vars,
  XYZArray &atomForceRec,
  XYZArray &molForceRec,
  const std::vector<double> &particleCharge,
  const std::vector<int> &particleMol,
  const std::vector<int> &particleKind,
  const std::vector<bool> &particleHasNoCharge,
  const std::vector<int> &startMol,
  const std::vector<int> &lengthMol,
  double alpha,
  double alphaSq,
  double qqFact,
  double constValue,
  uint imageSize,
  XYZArray const &molCoords,
  int boxStart,
  int boxEnd,
  BoxDimensions const &boxAxes, 
  int box
)
{
  int numberOfAtomsInsideBox = boxEnd - boxStart;
  int atomCount = atomForceRec.Count();
  int molCount = molForceRec.Count();
  double *gpu_particleCharge;
  int *gpu_particleMol, *gpu_particleKind;
  bool *gpu_particleHasNoCharge;
  bool *arr_particleHasNoCharge = new bool[particleHasNoCharge.size()];
  int *gpu_startMol, *gpu_lengthMol;

  // particleHasNoCharge is stored in vector<bool>, so in order to copy it to GPU
  // it needs to be stored in bool[]. because:
  // std::vector<bool> : Does not necessarily store its elements as a contiguous array
  for(int i=0; i<particleHasNoCharge.size(); i++) {
    arr_particleHasNoCharge[i] = particleHasNoCharge[i];
  }

  // calculate block and grid sizes
  int threadsPerBlock = 256;
  int blocksPerGrid = numberOfAtomsInsideBox;

  CUMALLOC((void **) &gpu_particleCharge, particleCharge.size() * sizeof(double));
  CUMALLOC((void **) &gpu_particleHasNoCharge, particleHasNoCharge.size() * sizeof(bool));
  CUMALLOC((void **) &gpu_startMol, startMol.size() * sizeof(int));
  CUMALLOC((void **) &gpu_lengthMol, lengthMol.size() * sizeof(int));
  CUMALLOC((void **) &gpu_particleMol, particleMol.size() * sizeof(int));
  CUMALLOC((void **) &gpu_particleKind, particleKind.size() * sizeof(int));

  cudaMemcpy(vars->gpu_aForceRecx, atomForceRec.x, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_aForceRecy, atomForceRec.y, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_aForceRecz, atomForceRec.z, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecx, molForceRec.x, sizeof(double) * molCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecy, molForceRec.y, sizeof(double) * molCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecz, molForceRec.z, sizeof(double) * molCount, cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0], sizeof(double) * particleCharge.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], sizeof(int) * particleMol.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0], sizeof(int) * particleKind.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleHasNoCharge, arr_particleHasNoCharge, sizeof(bool) * particleHasNoCharge.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, molCoords.x, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, molCoords.y, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, molCoords.z, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_startMol, &startMol[0], sizeof(int) * startMol.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_lengthMol, &lengthMol[0], sizeof(int) * lengthMol.size(), cudaMemcpyHostToDevice);

  checkLastErrorCUDA(__FILE__, __LINE__);
  BoxForceReciprocalGPU<<<blocksPerGrid, threadsPerBlock>>>(
    vars->gpu_aForceRecx,
    vars->gpu_aForceRecy,
    vars->gpu_aForceRecz,
    vars->gpu_mForceRecx,
    vars->gpu_mForceRecy,
    vars->gpu_mForceRecz,
    gpu_particleCharge,
    gpu_particleMol,
    gpu_particleKind,
    gpu_particleHasNoCharge,
    gpu_startMol,
    gpu_lengthMol,
    alpha,
    alphaSq,
    qqFact,
    constValue,
    imageSize,
    vars->gpu_kx[box],
    vars->gpu_ky[box],
    vars->gpu_kz[box],
    vars->gpu_x,
    vars->gpu_y,
    vars->gpu_z,
    vars->gpu_prefact[box],
    vars->gpu_sumRnew[box],
    vars->gpu_sumInew[box],
    vars->gpu_isFraction,
    vars->gpu_molIndex,
    vars->gpu_kindIndex,
    vars->gpu_lambdaCoulomb,
    boxAxes.GetAxis(box).x,
    boxAxes.GetAxis(box).y,
    boxAxes.GetAxis(box).z,
    box
  );
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  cudaMemcpy(atomForceRec.x, vars->gpu_aForceRecx, sizeof(double) * atomCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(atomForceRec.y, vars->gpu_aForceRecy, sizeof(double) * atomCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(atomForceRec.z, vars->gpu_aForceRecz, sizeof(double) * atomCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(molForceRec.x, vars->gpu_mForceRecx, sizeof(double) * molCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(molForceRec.y, vars->gpu_mForceRecy, sizeof(double) * molCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(molForceRec.z, vars->gpu_mForceRecz, sizeof(double) * molCount, cudaMemcpyDeviceToHost);

  cudaDeviceSynchronize();
  delete [] arr_particleHasNoCharge;
  CUFREE(gpu_particleCharge);
  CUFREE(gpu_particleHasNoCharge);
  CUFREE(gpu_startMol);
  CUFREE(gpu_lengthMol);
  CUFREE(gpu_particleMol);
  CUFREE(gpu_particleKind);
}

__global__ void BoxForceReciprocalGPU(
  double *gpu_aForceRecx,
  double *gpu_aForceRecy,
  double *gpu_aForceRecz,
  double *gpu_mForceRecx,
  double *gpu_mForceRecy,
  double *gpu_mForceRecz,
  double *gpu_particleCharge,
  int *gpu_particleMol,
  int *gpu_particleKind,
  bool *gpu_particleHasNoCharge,
  int *gpu_startMol,
  int *gpu_lengthMol,
  double alpha,
  double alphaSq,
  double qqFact,
  double constValue,
  int imageSize,
  double *gpu_kx,
  double *gpu_ky,
  double *gpu_kz,
  double *gpu_x,
  double *gpu_y,
  double *gpu_z,
  double *gpu_prefact,
  double *gpu_sumRnew,
  double *gpu_sumInew,
  bool *gpu_isFraction,
  int *gpu_molIndex,
  int *gpu_kindIndex,
  double *gpu_lambdaCoulomb,
  double axx,
  double axy,
  double axz,
  int box
)
{
  __shared__ double shared[24];
  int laneID = threadIdx.x % 32;
  int warpID = threadIdx.x / 32;
  int particleID = blockIdx.x;
  double forceX = 0.0, forceY = 0.0, forceZ = 0.0;
  int moleculeID = gpu_particleMol[particleID];
  int kindID = gpu_particleKind[particleID];
  if(!gpu_particleHasNoCharge[particleID]) {
    double lambdaCoef = DeviceGetLambdaCoulomb(moleculeID, kindID, box, gpu_isFraction, gpu_molIndex, gpu_kindIndex, gpu_lambdaCoulomb);

    // loop over other particles within the same molecule
    if(threadIdx.x == 0) {
      double intraForce = 0.0, distSq = 0.0, dist = 0.0;
      double distVectX = 0.0, distVectY = 0.0, distVectZ = 0.0;
      int lastParticleWithinSameMolecule = gpu_startMol[particleID] + gpu_lengthMol[particleID];
      for(int otherParticle = gpu_startMol[particleID];
        otherParticle < lastParticleWithinSameMolecule;
        otherParticle++)
      {
        if(particleID != otherParticle) {
          DeviceInRcut(distSq, distVectX, distVectY, distVectZ, gpu_x, gpu_y, gpu_z, particleID, otherParticle, axx, axy, axz, box);
          dist = sqrt(distSq);

          double expConstValue = exp(-1.0 * alphaSq * distSq);
          double qiqj = gpu_particleCharge[particleID] * gpu_particleCharge[otherParticle] * qqFact;
          intraForce = qiqj * lambdaCoef * lambdaCoef / distSq;
          intraForce *= ((erf(alpha * dist) / dist) - constValue * expConstValue);
          forceX -= intraForce * distVectX;
          forceY -= intraForce * distVectY;
          forceZ -= intraForce * distVectZ;
        }
      }
    }

    // loop over images
    for(int vectorIndex = threadIdx.x; vectorIndex < imageSize; vectorIndex += blockDim.x) {
      double dot = gpu_x[particleID] * gpu_kx[vectorIndex] +
        gpu_y[particleID] * gpu_ky[vectorIndex] + 
        gpu_z[particleID] * gpu_kz[vectorIndex];
        
      double factor = 2.0 * gpu_particleCharge[particleID] * gpu_prefact[vectorIndex] * lambdaCoef *
        (sin(dot) * gpu_sumRnew[vectorIndex] - cos(dot) * gpu_sumInew[vectorIndex]);
        
      forceX += factor * gpu_kx[vectorIndex];
      forceY += factor * gpu_ky[vectorIndex];
      forceZ += factor * gpu_kz[vectorIndex];
    }
  }

  // perform reduction at this point
  int warpSize = 32;
  for (int offset = warpSize/2; offset > 0; offset /= 2) {
    forceX += __shfl_down_sync(FULL_MASK, forceX, offset);
    forceY += __shfl_down_sync(FULL_MASK, forceY, offset);
    forceZ += __shfl_down_sync(FULL_MASK, forceZ, offset);
  }
  if(laneID == 0) {
    shared[warpID*3+0] = forceX;
    shared[warpID*3+1] = forceY;
    shared[warpID*3+2] = forceZ;
  }

  // first thread inside the block will write back to global memory
  __syncthreads();
  if(threadIdx.x == 0) {
    for(int w=1; w<8; w++) {
      forceX += shared[w*3+0];
      forceY += shared[w*3+1];
      forceZ += shared[w*3+2];
    }
    gpu_aForceRecx[particleID] = forceX;
    gpu_aForceRecy[particleID] = forceY;
    gpu_aForceRecz[particleID] = forceZ;
    atomicAdd(&gpu_mForceRecx[moleculeID], forceX);
    atomicAdd(&gpu_mForceRecy[moleculeID], forceY);
    atomicAdd(&gpu_mForceRecz[moleculeID], forceZ);
  }
}

__global__ void SwapReciprocalGPU(double *gpu_x, double *gpu_y, double *gpu_z,
                                  double *gpu_kx, double *gpu_ky, double *gpu_kz,
                                  int atomNumber,
                                  double *gpu_particleCharge,
                                  double *gpu_sumRnew,
                                  double *gpu_sumInew,
                                  double *gpu_sumRref,
                                  double *gpu_sumIref,
                                  double *gpu_prefactRef,
                                  int insert,
                                  double *gpu_energyRecipNew,
                                  int imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  int p;
  double dotProduct = 0.0, sumReal = 0.0, sumImaginary = 0.0;

  for(p = 0; p < atomNumber; p++) {
    dotProduct = DotProductGPU(gpu_kx[threadID], gpu_ky[threadID],
                               gpu_kz[threadID], gpu_x[p], gpu_y[p], gpu_z[p]);
    sumReal += (gpu_particleCharge[p] * cos(dotProduct));
    sumImaginary += (gpu_particleCharge[p] * sin(dotProduct));
  }

  //If we insert the molecule to the box, we add the sum value.
  //Otherwise, we subtract the sum value
  if(insert) {
    gpu_sumRnew[threadID] = gpu_sumRref[threadID] + sumReal;
    gpu_sumInew[threadID] = gpu_sumIref[threadID] + sumImaginary;
  } else {
    gpu_sumRnew[threadID] = gpu_sumRref[threadID] - sumReal;
    gpu_sumInew[threadID] = gpu_sumIref[threadID] - sumImaginary;
  }

  gpu_energyRecipNew[threadID] = ((gpu_sumRnew[threadID] *
                                   gpu_sumRnew[threadID] +
                                   gpu_sumInew[threadID] *
                                   gpu_sumInew[threadID]) *
                                  gpu_prefactRef[threadID]);
}

__global__ void MolReciprocalGPU(double *gpu_cx, double *gpu_cy, double *gpu_cz,
                                 double *gpu_nx, double *gpu_ny, double *gpu_nz,
                                 double *gpu_kx, double *gpu_ky, double *gpu_kz,
                                 int atomNumber,
                                 double *gpu_particleCharge,
                                 double *gpu_sumRnew,
                                 double *gpu_sumInew,
                                 double *gpu_sumRref,
                                 double *gpu_sumIref,
                                 double *gpu_prefactRef,
                                 double *gpu_energyRecipNew,
                                 int imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;
  int p;
  double dotProductOld = 0.0, dotProductNew = 0.0;
  double sumRealNew = 0.0, sumImaginaryNew = 0.0;
  double sumRealOld = 0.0, sumImaginaryOld = 0.0;

  for(p = 0; p < atomNumber; p++) {
    dotProductOld = DotProductGPU(gpu_kx[threadID], gpu_ky[threadID],
                                  gpu_kz[threadID],
                                  gpu_cx[p], gpu_cy[p], gpu_cz[p]);
    dotProductNew = DotProductGPU(gpu_kx[threadID], gpu_ky[threadID],
                                  gpu_kz[threadID],
                                  gpu_nx[p], gpu_ny[p], gpu_nz[p]);
    sumRealNew += (gpu_particleCharge[p] * cos(dotProductNew));
    sumImaginaryNew += (gpu_particleCharge[p] * sin(dotProductNew));
    sumRealOld += (gpu_particleCharge[p] * cos(dotProductOld));
    sumImaginaryOld += (gpu_particleCharge[p] * sin(dotProductOld));
  }

  gpu_sumRnew[threadID] = gpu_sumRref[threadID] - sumRealOld + sumRealNew;
  gpu_sumInew[threadID] = gpu_sumIref[threadID] - sumImaginaryOld +
                          sumImaginaryNew;

  gpu_energyRecipNew[threadID] = ((gpu_sumRnew[threadID] *
                                   gpu_sumRnew[threadID] +
                                   gpu_sumInew[threadID] *
                                   gpu_sumInew[threadID]) *
                                  gpu_prefactRef[threadID]);
}

__global__ void BoxReciprocalSetupGPU(double *gpu_x,
                                      double *gpu_y,
                                      double *gpu_z,
                                      double *gpu_kx,
                                      double *gpu_ky,
                                      double *gpu_kz,
                                      double atomNumber,
                                      double *gpu_particleCharge,
                                      double *gpu_sumRnew,
                                      double *gpu_sumInew,
                                      int imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;
  int i;
  double dotP;

  gpu_sumRnew[threadID] = 0.0;
  gpu_sumInew[threadID] = 0.0;
  for(i = 0; i < atomNumber; i++) {
    dotP = DotProductGPU(gpu_kx[threadID], gpu_ky[threadID], gpu_kz[threadID],
                         gpu_x[i], gpu_y[i], gpu_z[i]);
    gpu_sumRnew[threadID] += gpu_particleCharge[i] * cos(dotP);
    gpu_sumInew[threadID] += gpu_particleCharge[i] * sin(dotP);
  }
}

__global__ void BoxReciprocalGPU(double *gpu_prefact,
                                 double *gpu_sumRnew,
                                 double *gpu_sumInew,
                                 double *gpu_energyRecip,
                                 int imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  gpu_energyRecip[threadID] = ((gpu_sumRnew[threadID] * gpu_sumRnew[threadID] +
                                gpu_sumInew[threadID] * gpu_sumInew[threadID]) *
                               gpu_prefact[threadID]);
}

#endif
