
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.70
Copyright (C) 2018  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA

#include <cuda.h>
#include <cuda_runtime.h>
#include "BoxDimensions.h"
#include "CalculateEwaldCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "CUDAMemoryManager.cuh"
#include "cub/cub.cuh"
#include <vector>

using namespace cub;

#define IMAGES_PER_BLOCK 64
#define PARTICLE_PER_BLOCK 64

#define FULL_MASK 0xffffffff

void CallBoxReciprocalSetupGPU(VariablesCUDA *vars,
                               XYZArray const &coords,
                               double const *kx,
                               double const *ky,
                               double const *kz,
                               std::vector<double> particleCharge,
                               uint imageSize,
                               double *sumRnew,
                               double *sumInew,
                               double *prefact,
                               double *hsqr,
                               double &energyRecip,
                               uint box)
{
  double *gpu_particleCharge;
  double * gpu_energyRecip;
  double * gpu_final_energyRecip;
  int atomNumber = coords.Count();

  CUMALLOC((void**) &gpu_particleCharge,
           particleCharge.size() * sizeof(double));
  CUMALLOC((void**) &gpu_energyRecip, imageSize * sizeof(double));
  CUMALLOC((void**) &gpu_final_energyRecip, sizeof(double));

  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_kx[box], kx, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_ky[box], ky, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_kz[box], kz, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_prefact[box], prefact, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemcpy(vars->gpu_hsqr[box], hsqr, imageSize * sizeof(double),
             cudaMemcpyHostToDevice);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemset(vars->gpu_sumRnew[box], 0, imageSize * sizeof(double));
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaMemset(vars->gpu_sumInew[box], 0, imageSize * sizeof(double));
  checkLastErrorCUDA(__FILE__, __LINE__);

  dim3 threadsPerBlock(256, 1, 1);
  dim3 blocksPerGrid((int)(imageSize / threadsPerBlock.x) + 1, (int)(atomNumber / PARTICLE_PER_BLOCK) + 1, 1);
  BoxReciprocalSetupGPU <<< blocksPerGrid, threadsPerBlock>>>(
    vars->gpu_x,
    vars->gpu_y,
    vars->gpu_z,
    vars->gpu_kx[box],
    vars->gpu_ky[box],
    vars->gpu_kz[box],
    atomNumber,
    gpu_particleCharge,
    vars->gpu_sumRnew[box],
    vars->gpu_sumInew[box],
    imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  //Need just one thread per image for this kernel.
  blocksPerGrid.y = 1;
  BoxReciprocalGPU <<< blocksPerGrid, threadsPerBlock>>>(
    vars->gpu_prefact[box],
    vars->gpu_sumRnew[box],
    vars->gpu_sumInew[box],
    gpu_energyRecip,
    imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  cudaMemcpy(sumRnew, vars->gpu_sumRnew[box],
             imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
  cudaMemcpy(sumInew, vars->gpu_sumInew[box],
             imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);

  // ReduceSum
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecip,
                    gpu_final_energyRecip, imageSize);
  CUMALLOC(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecip,
                    gpu_final_energyRecip, imageSize);
  cudaMemcpy(&energyRecip, gpu_final_energyRecip,
             sizeof(double), cudaMemcpyDeviceToHost);

  CUFREE(gpu_particleCharge);
  CUFREE(gpu_energyRecip);
  CUFREE(gpu_final_energyRecip);
  CUFREE(d_temp_storage);
}

__global__ void BoxReciprocalSetupGPU(double *gpu_x,
                                      double *gpu_y,
                                      double *gpu_z,
                                      double *gpu_kx,
                                      double *gpu_ky,
                                      double *gpu_kz,
                                      int atomNumber,
                                      double *gpu_particleCharge,
                                      double *gpu_sumRnew,
                                      double *gpu_sumInew,
                                      int imageSize)
{
  __shared__ double shared_coords[PARTICLE_PER_BLOCK * 3];
  int imageID = blockIdx.x * blockDim.x + threadIdx.x;
  int offset_coordinates_index = blockIdx.y * PARTICLE_PER_BLOCK;
  int numberOfAtoms = min(PARTICLE_PER_BLOCK, atomNumber - offset_coordinates_index);
  double sumR = 0.0, sumI = 0.0;

  if(threadIdx.x < numberOfAtoms) {
    shared_coords[threadIdx.x * 3    ] = gpu_x[offset_coordinates_index + threadIdx.x];
    shared_coords[threadIdx.x * 3 + 1] = gpu_y[offset_coordinates_index + threadIdx.x];
    shared_coords[threadIdx.x * 3 + 2] = gpu_z[offset_coordinates_index + threadIdx.x];
  }

  if(imageID >= imageSize)
    return;

  __syncthreads();
  for(int particleID = 0; particleID < numberOfAtoms; particleID++) {
    double dot = DotProductGPU(gpu_kx[imageID], gpu_ky[imageID], gpu_kz[imageID],
                               shared_coords[particleID * 3], shared_coords[particleID * 3 + 1],
                               shared_coords[particleID * 3 + 2]);
    double dotsin, dotcos;
    sincos(dot, &dotsin, &dotcos);
    sumR += gpu_particleCharge[offset_coordinates_index + particleID] * dotcos;
    sumI += gpu_particleCharge[offset_coordinates_index + particleID] * dotsin;
  }

  atomicAdd(&gpu_sumRnew[imageID], sumR);
  atomicAdd(&gpu_sumInew[imageID], sumI);
}

__global__ void BoxReciprocalGPU(double *gpu_prefact,
                                 double *gpu_sumRnew,
                                 double *gpu_sumInew,
                                 double *gpu_energyRecip,
                                 int imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  gpu_energyRecip[threadID] = ((gpu_sumRnew[threadID] * gpu_sumRnew[threadID] +
                                gpu_sumInew[threadID] * gpu_sumInew[threadID]) *
                               gpu_prefact[threadID]);
}

void CallMolReciprocalGPU(VariablesCUDA *vars,
                          XYZArray const &currentCoords,
                          XYZArray const &newCoords,
                          std::vector<double> particleCharge,
                          uint imageSize,
                          double *sumRnew,
                          double *sumInew,
                          double &energyRecipNew,
                          uint box)
{
  // Calculate atom number
  int atomNumber = currentCoords.Count();
  int newCoordsNumber = newCoords.Count();
  double *gpu_particleCharge;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_energyRecipNew, *gpu_final_energyRecipNew;

  CUMALLOC((void**) &gpu_particleCharge,
           particleCharge.size() * sizeof(double));
  CUMALLOC((void**) &gpu_energyRecipNew, imageSize * sizeof(double));
  CUMALLOC((void**) &gpu_final_energyRecipNew, sizeof(double));

  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_nx, newCoords.x, newCoordsNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_ny, newCoords.y, newCoordsNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_nz, newCoords.z, newCoordsNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  MolReciprocalGPU <<< blocksPerGrid,
                   threadsPerBlock>>>(vars->gpu_x, vars->gpu_y, vars->gpu_z,
                                      vars->gpu_nx, vars->gpu_ny, vars->gpu_nz,
                                      vars->gpu_kxRef[box], vars->gpu_kyRef[box],
                                      vars->gpu_kzRef[box],
                                      atomNumber,
                                      gpu_particleCharge,
                                      vars->gpu_sumRnew[box],
                                      vars->gpu_sumInew[box],
                                      vars->gpu_sumRref[box],
                                      vars->gpu_sumIref[box],
                                      vars->gpu_prefactRef[box],
                                      gpu_energyRecipNew,
                                      imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  cudaMemcpy(sumRnew, vars->gpu_sumRnew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
  cudaMemcpy(sumInew, vars->gpu_sumInew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);

  // ReduceSum
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  CUMALLOC(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  cudaMemcpy(&energyRecipNew, gpu_final_energyRecipNew,
             sizeof(double), cudaMemcpyDeviceToHost);


  CUFREE(gpu_particleCharge);
  CUFREE(gpu_energyRecipNew);
  CUFREE(gpu_final_energyRecipNew);
  CUFREE(d_temp_storage);
}

void CallSwapReciprocalGPU(VariablesCUDA *vars,
                           XYZArray const &coords,
                           std::vector<double> particleCharge,
                           uint imageSize,
                           double *sumRnew,
                           double *sumInew,
                           int const insert,
                           double &energyRecipNew,
                           uint box)
{
  // Calculate atom number
  int atomNumber = coords.Count();
  // given coordinates
  double *gpu_particleCharge;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_energyRecipNew, *gpu_final_energyRecipNew;

  CUMALLOC((void**) &gpu_particleCharge,
           particleCharge.size() * sizeof(double));
  CUMALLOC((void**) &gpu_energyRecipNew, imageSize * sizeof(double));
  CUMALLOC((void**) &gpu_final_energyRecipNew, sizeof(double));

  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  SwapReciprocalGPU <<< blocksPerGrid,
                    threadsPerBlock>>>(vars->gpu_x, vars->gpu_y, vars->gpu_z,
                                       vars->gpu_kxRef[box], vars->gpu_kyRef[box],
                                       vars->gpu_kzRef[box],
                                       atomNumber,
                                       gpu_particleCharge,
                                       vars->gpu_sumRnew[box],
                                       vars->gpu_sumInew[box],
                                       vars->gpu_sumRref[box],
                                       vars->gpu_sumIref[box],
                                       vars->gpu_prefactRef[box],
                                       insert,
                                       gpu_energyRecipNew,
                                       imageSize);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);
//#ifndef NDEBUG
  // In the future maybe we could remove this for Nondebug?
  cudaMemcpy(sumRnew, vars->gpu_sumRnew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
  cudaMemcpy(sumInew, vars->gpu_sumInew[box], imageSize * sizeof(double),
             cudaMemcpyDeviceToHost);
//#endif

  // ReduceSum
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  CUMALLOC(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_energyRecipNew,
                    gpu_final_energyRecipNew, imageSize);
  cudaMemcpy(&energyRecipNew, gpu_final_energyRecipNew,
             sizeof(double), cudaMemcpyDeviceToHost);

  CUFREE(gpu_particleCharge);
  CUFREE(gpu_energyRecipNew);
  CUFREE(gpu_final_energyRecipNew);
  CUFREE(d_temp_storage);
}

void CallBoxForceReciprocalGPU(
  VariablesCUDA *vars,
  XYZArray &atomForceRec,
  XYZArray &molForceRec,
  const std::vector<double> &particleCharge,
  const std::vector<int> &particleMol,
  const std::vector<int> &particleKind,
  const std::vector<bool> &particleHasNoCharge,
  const std::vector<int> &startMol,
  const std::vector<int> &lengthMol,
  double alpha,
  double alphaSq,
  double qqFact,
  double constValue,
  uint imageSize,
  XYZArray const &molCoords,
  int boxStart,
  int boxEnd,
  BoxDimensions const &boxAxes,
  int box
)
{
  int numberOfAtomsInsideBox = boxEnd - boxStart;
  int atomCount = atomForceRec.Count();
  int molCount = molForceRec.Count();
  double *gpu_particleCharge;
  int *gpu_particleMol, *gpu_particleKind;
  bool *gpu_particleHasNoCharge;
  bool *arr_particleHasNoCharge = new bool[particleHasNoCharge.size()];
  int *gpu_startMol, *gpu_lengthMol;

  // particleHasNoCharge is stored in vector<bool>, so in order to copy it to GPU
  // it needs to be stored in bool[]. because:
  // std::vector<bool> : Does not necessarily store its elements as a contiguous array
  for(int i = 0; i < particleHasNoCharge.size(); i++) {
    arr_particleHasNoCharge[i] = particleHasNoCharge[i];
  }

  // calculate block and grid sizes
  dim3 threadsPerBlock(256, 1, 1);
  int blocksPerGridX = (int)(numberOfAtomsInsideBox / threadsPerBlock.x) + 1;
  int blocksPerGridY = (int)(imageSize / IMAGES_PER_BLOCK) + 1;
  dim3 blocksPerGrid(blocksPerGridX, blocksPerGridY, 1);

  CUMALLOC((void **) &gpu_particleCharge, particleCharge.size() * sizeof(double));
  CUMALLOC((void **) &gpu_particleHasNoCharge, particleHasNoCharge.size() * sizeof(bool));
  CUMALLOC((void **) &gpu_startMol, startMol.size() * sizeof(int));
  CUMALLOC((void **) &gpu_lengthMol, lengthMol.size() * sizeof(int));
  CUMALLOC((void **) &gpu_particleMol, particleMol.size() * sizeof(int));
  CUMALLOC((void **) &gpu_particleKind, particleKind.size() * sizeof(int));

  cudaMemcpy(vars->gpu_aForceRecx, atomForceRec.x, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_aForceRecy, atomForceRec.y, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_aForceRecz, atomForceRec.z, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecx, molForceRec.x, sizeof(double) * molCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecy, molForceRec.y, sizeof(double) * molCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecz, molForceRec.z, sizeof(double) * molCount, cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0], sizeof(double) * particleCharge.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], sizeof(int) * particleMol.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0], sizeof(int) * particleKind.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleHasNoCharge, arr_particleHasNoCharge, sizeof(bool) * particleHasNoCharge.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, molCoords.x, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, molCoords.y, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, molCoords.z, sizeof(double) * atomCount, cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_startMol, &startMol[0], sizeof(int) * startMol.size(), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_lengthMol, &lengthMol[0], sizeof(int) * lengthMol.size(), cudaMemcpyHostToDevice);

  checkLastErrorCUDA(__FILE__, __LINE__);
  BoxForceReciprocalGPU <<< blocksPerGrid, threadsPerBlock>>>(
    vars->gpu_aForceRecx,
    vars->gpu_aForceRecy,
    vars->gpu_aForceRecz,
    vars->gpu_mForceRecx,
    vars->gpu_mForceRecy,
    vars->gpu_mForceRecz,
    gpu_particleCharge,
    gpu_particleMol,
    gpu_particleKind,
    gpu_particleHasNoCharge,
    gpu_startMol,
    gpu_lengthMol,
    alpha,
    alphaSq,
    qqFact,
    constValue,
    imageSize,
    vars->gpu_kx[box],
    vars->gpu_ky[box],
    vars->gpu_kz[box],
    vars->gpu_x,
    vars->gpu_y,
    vars->gpu_z,
    vars->gpu_prefact[box],
    vars->gpu_sumRnew[box],
    vars->gpu_sumInew[box],
    vars->gpu_isFraction,
    vars->gpu_molIndex,
    vars->gpu_kindIndex,
    vars->gpu_lambdaCoulomb,
    boxAxes.GetAxis(box).x,
    boxAxes.GetAxis(box).y,
    boxAxes.GetAxis(box).z,
    box,
    numberOfAtomsInsideBox
  );
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  cudaMemcpy(atomForceRec.x, vars->gpu_aForceRecx, sizeof(double) * atomCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(atomForceRec.y, vars->gpu_aForceRecy, sizeof(double) * atomCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(atomForceRec.z, vars->gpu_aForceRecz, sizeof(double) * atomCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(molForceRec.x, vars->gpu_mForceRecx, sizeof(double) * molCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(molForceRec.y, vars->gpu_mForceRecy, sizeof(double) * molCount, cudaMemcpyDeviceToHost);
  cudaMemcpy(molForceRec.z, vars->gpu_mForceRecz, sizeof(double) * molCount, cudaMemcpyDeviceToHost);

  cudaDeviceSynchronize();
  delete [] arr_particleHasNoCharge;
  CUFREE(gpu_particleCharge);
  CUFREE(gpu_particleHasNoCharge);
  CUFREE(gpu_startMol);
  CUFREE(gpu_lengthMol);
  CUFREE(gpu_particleMol);
  CUFREE(gpu_particleKind);
}

__global__ void BoxForceReciprocalGPU(
  double *gpu_aForceRecx,
  double *gpu_aForceRecy,
  double *gpu_aForceRecz,
  double *gpu_mForceRecx,
  double *gpu_mForceRecy,
  double *gpu_mForceRecz,
  double *gpu_particleCharge,
  int *gpu_particleMol,
  int *gpu_particleKind,
  bool *gpu_particleHasNoCharge,
  int *gpu_startMol,
  int *gpu_lengthMol,
  double alpha,
  double alphaSq,
  double qqFact,
  double constValue,
  int imageSize,
  double *gpu_kx,
  double *gpu_ky,
  double *gpu_kz,
  double *gpu_x,
  double *gpu_y,
  double *gpu_z,
  double *gpu_prefact,
  double *gpu_sumRnew,
  double *gpu_sumInew,
  bool *gpu_isFraction,
  int *gpu_molIndex,
  int *gpu_kindIndex,
  double *gpu_lambdaCoulomb,
  double axx,
  double axy,
  double axz,
  int box,
  int numberOfAtomsInsideBox
)
{
  __shared__ double shared_kvector[IMAGES_PER_BLOCK * 3];
  int particleID =  blockDim.x * blockIdx.x + threadIdx.x;
  int offset_vector_index = blockIdx.y * IMAGES_PER_BLOCK;
  int numberOfVectors = min(IMAGES_PER_BLOCK, imageSize - offset_vector_index);

  if(threadIdx.x < numberOfVectors) {
    shared_kvector[threadIdx.x * 3] = gpu_kx[offset_vector_index + threadIdx.x];
    shared_kvector[threadIdx.x * 3 + 1] = gpu_ky[offset_vector_index + threadIdx.x];
    shared_kvector[threadIdx.x * 3 + 2] = gpu_kz[offset_vector_index + threadIdx.x];
  }

  if (particleID >= numberOfAtomsInsideBox) return;
  double forceX = 0.0, forceY = 0.0, forceZ = 0.0;
  int moleculeID = gpu_particleMol[particleID];
  int kindID = gpu_particleKind[particleID];

  if(gpu_particleHasNoCharge[particleID])
    return;

  double x = gpu_x[particleID];
  double y = gpu_y[particleID];
  double z = gpu_z[particleID];
  double lambdaCoef = DeviceGetLambdaCoulomb(moleculeID, kindID, box, gpu_isFraction, gpu_molIndex, gpu_kindIndex, gpu_lambdaCoulomb);

  __syncthreads();
  // loop over images
  for(int vectorIndex = 0; vectorIndex < numberOfVectors; vectorIndex ++) {
    double dot = x * shared_kvector[vectorIndex * 3] + y *
                 shared_kvector[vectorIndex * 3 + 1] + z * shared_kvector[vectorIndex * 3 + 2];
    double dotsin, dotcos;
    sincos(dot, &dotsin, &dotcos);
    double factor = 2.0 * gpu_particleCharge[particleID] *
                    gpu_prefact[offset_vector_index + vectorIndex] * lambdaCoef *
                    (dotsin * gpu_sumRnew[offset_vector_index + vectorIndex] -
                     dotcos * gpu_sumInew[offset_vector_index + vectorIndex]);

    forceX += factor * shared_kvector[vectorIndex * 3];
    forceY += factor * shared_kvector[vectorIndex * 3 + 1];
    forceZ += factor * shared_kvector[vectorIndex * 3 + 2];
  }

  // loop over other particles within the same molecule
  if(blockIdx.y == 0) {
    double intraForce = 0.0, distSq = 0.0, dist = 0.0;
    double distVectX = 0.0, distVectY = 0.0, distVectZ = 0.0;
    int lastParticleWithinSameMolecule = gpu_startMol[particleID] + gpu_lengthMol[particleID];
    for(int otherParticle = gpu_startMol[particleID];
        otherParticle < lastParticleWithinSameMolecule;
        otherParticle++) {
      if(particleID != otherParticle) {
        DeviceInRcut(distSq, distVectX, distVectY, distVectZ, gpu_x, gpu_y, gpu_z, particleID, otherParticle, axx, axy, axz, box);
        dist = sqrt(distSq);

        double expConstValue = exp(-1.0 * alphaSq * distSq);
        double qiqj = gpu_particleCharge[particleID] * gpu_particleCharge[otherParticle] * qqFact;
        intraForce = qiqj * lambdaCoef * lambdaCoef / distSq;
        intraForce *= ((erf(alpha * dist) / dist) - constValue * expConstValue);
        forceX -= intraForce * distVectX;
        forceY -= intraForce * distVectY;
        forceZ -= intraForce * distVectZ;
      }
    }
  }

  atomicAdd(&gpu_aForceRecx[particleID], forceX);
  atomicAdd(&gpu_aForceRecy[particleID], forceY);
  atomicAdd(&gpu_aForceRecz[particleID], forceZ);
  atomicAdd(&gpu_mForceRecx[moleculeID], forceX);
  atomicAdd(&gpu_mForceRecy[moleculeID], forceY);
  atomicAdd(&gpu_mForceRecz[moleculeID], forceZ);

}

__global__ void SwapReciprocalGPU(double *gpu_x, double *gpu_y, double *gpu_z,
                                  double *gpu_kx, double *gpu_ky, double *gpu_kz,
                                  int atomNumber,
                                  double *gpu_particleCharge,
                                  double *gpu_sumRnew,
                                  double *gpu_sumInew,
                                  double *gpu_sumRref,
                                  double *gpu_sumIref,
                                  double *gpu_prefactRef,
                                  int insert,
                                  double *gpu_energyRecipNew,
                                  int imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  int p;
  double dotProduct = 0.0, sumReal = 0.0, sumImaginary = 0.0;

  for(p = 0; p < atomNumber; p++) {
    dotProduct = DotProductGPU(gpu_kx[threadID], gpu_ky[threadID],
                               gpu_kz[threadID], gpu_x[p], gpu_y[p], gpu_z[p]);
    sumReal += (gpu_particleCharge[p] * cos(dotProduct));
    sumImaginary += (gpu_particleCharge[p] * sin(dotProduct));
  }

  //If we insert the molecule to the box, we add the sum value.
  //Otherwise, we subtract the sum value
  if(insert) {
    gpu_sumRnew[threadID] = gpu_sumRref[threadID] + sumReal;
    gpu_sumInew[threadID] = gpu_sumIref[threadID] + sumImaginary;
  } else {
    gpu_sumRnew[threadID] = gpu_sumRref[threadID] - sumReal;
    gpu_sumInew[threadID] = gpu_sumIref[threadID] - sumImaginary;
  }

  gpu_energyRecipNew[threadID] = ((gpu_sumRnew[threadID] *
                                   gpu_sumRnew[threadID] +
                                   gpu_sumInew[threadID] *
                                   gpu_sumInew[threadID]) *
                                  gpu_prefactRef[threadID]);
}

__global__ void MolReciprocalGPU(double *gpu_cx, double *gpu_cy, double *gpu_cz,
                                 double *gpu_nx, double *gpu_ny, double *gpu_nz,
                                 double *gpu_kx, double *gpu_ky, double *gpu_kz,
                                 int atomNumber,
                                 double *gpu_particleCharge,
                                 double *gpu_sumRnew,
                                 double *gpu_sumInew,
                                 double *gpu_sumRref,
                                 double *gpu_sumIref,
                                 double *gpu_prefactRef,
                                 double *gpu_energyRecipNew,
                                 int imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;
  int p;
  double dotProductOld = 0.0, dotProductNew = 0.0;
  double sumRealNew = 0.0, sumImaginaryNew = 0.0;
  double sumRealOld = 0.0, sumImaginaryOld = 0.0;

  for(p = 0; p < atomNumber; p++) {
    dotProductOld = DotProductGPU(gpu_kx[threadID], gpu_ky[threadID],
                                  gpu_kz[threadID],
                                  gpu_cx[p], gpu_cy[p], gpu_cz[p]);
    dotProductNew = DotProductGPU(gpu_kx[threadID], gpu_ky[threadID],
                                  gpu_kz[threadID],
                                  gpu_nx[p], gpu_ny[p], gpu_nz[p]);
    sumRealNew += (gpu_particleCharge[p] * cos(dotProductNew));
    sumImaginaryNew += (gpu_particleCharge[p] * sin(dotProductNew));
    sumRealOld += (gpu_particleCharge[p] * cos(dotProductOld));
    sumImaginaryOld += (gpu_particleCharge[p] * sin(dotProductOld));
  }

  gpu_sumRnew[threadID] = gpu_sumRref[threadID] - sumRealOld + sumRealNew;
  gpu_sumInew[threadID] = gpu_sumIref[threadID] - sumImaginaryOld +
                          sumImaginaryNew;

  gpu_energyRecipNew[threadID] = ((gpu_sumRnew[threadID] *
                                   gpu_sumRnew[threadID] +
                                   gpu_sumInew[threadID] *
                                   gpu_sumInew[threadID]) *
                                  gpu_prefactRef[threadID]);
}

#endif
