#include "CUDAMemoryManager.cuh"

#ifdef GOMC_CUDA
long long CUDAMemoryManager::totalAllocatedBytes = 0;
std::unordered_map<void *, std::pair<unsigned int, std::string> > CUDAMemoryManager::allocatedPointers;


hipError_t CUDAMemoryManager::mallocMemory(void **address, unsigned int size, std::string var_name)
{
  hipError_t ret = hipMalloc(address, size);
  allocatedPointers[*address] = make_pair(size, var_name);
  totalAllocatedBytes += size;
  return ret;
}

hipError_t CUDAMemoryManager::freeMemory(void *address, std::string var_name)
{
  if(allocatedPointers.find(address) != allocatedPointers.end()) {
    totalAllocatedBytes -= allocatedPointers[address].first;
    allocatedPointers.erase(address);
  } else {
    std::cout << "Warning! You are trying to free " << var_name << " where it was freed\n" <<
              "\tor never been allocated before!\n";
  }
  return hipFree(address);
}

bool CUDAMemoryManager::isFreed()
{
  bool ret = allocatedPointers.size() == 0;
  while(allocatedPointers.size() != 0) {
    auto it = allocatedPointers.begin();
    std::cout << "You forgot to free memory " << it->second.second
              << " with " << it->second.first << " bytes allocated to it!\n";
    std::cout << "I am going to free it for you!\n";
    freeMemory(it->first, it->second.second);
  }
  return ret;
}

#endif
